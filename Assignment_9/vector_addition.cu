
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = num;
    }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *vector, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main()
{
    const int N = 2 << 24;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWith(1, a, N);
    initWith(1, b, N);
    initWith(0, c, N);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 128;
    numberOfBlocks = 32;

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
    hipDeviceSynchronize();

    checkElementsAre(2, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
