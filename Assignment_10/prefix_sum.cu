
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float val, float *arr, int N)
{
  for (int i = 0; i < N; i++)
  {
    arr[i] = val;
  }
}

__global__ void prefixSum(float *arr, float *res, float *ptemp, float *ttemp, int N)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = gridDim.x * blockDim.x;
  int elementsPerThread = ceil(1.0 * N / totalThreads);

  int start = threadId * elementsPerThread;
  int count = 0;
  float *sums = new float[elementsPerThread];
  float sum = 0;

  for (int i = start; i < N && count < elementsPerThread; i++, count++)
  {
    sum += arr[i];
    sums[count] = sum;
  }

  float localSum;
  if (count)
    localSum = sums[count - 1];
  else
    localSum = 0;
  ptemp[threadId] = localSum;
  ttemp[threadId] = localSum;

  __syncthreads();

  if (totalThreads == 1)
  {
    for (int i = 0; i < N; i++)
      res[i] = sums[i];
  }
  else
  {
    int d = 0; // log2(totalThreads)
    int x = totalThreads;
    while (x > 1)
    {
      d++;
      x = x >> 1;
    }

    x = 1;
    for (int i = 0; i < 2 * d; i++)
    {
      int tsum = ttemp[threadId];

      __syncthreads();

      int newId = threadId / x;
      if (newId % 2 == 0)
      {
        int nextId = threadId + x;
        ptemp[nextId] += tsum;
        ttemp[nextId] += tsum;
      }
      else
      {
        int nextId = threadId - x;
        ttemp[nextId] += tsum;
      }

      x = x << 1;
    }

    __syncthreads();

    float diff = ptemp[threadId] - localSum;
    for (int i = start, j = 0; i < N && j < count; i++, j++)
    {
      res[i] = sums[j] + diff;
    }
  }
}

void checkRes(float *arr, float *res, int N)
{
  float sum = 0;
  for (int i = 0; i < N; i++)
  {
    sum += arr[i];
    if (sum != res[i])
    {
      printf("FAIL: res[%d] - %0.0f does not equal %0.0f\n", i, res[i], sum);
      exit(1);
    }
  }
  printf("SUCCESS! All prefix sums added correctly.\n");
}

int main()
{
  const int N = 1000000;
  size_t size = N * sizeof(float);

  float *arr;
  float *res;

  hipMallocManaged(&arr, size);
  hipMallocManaged(&res, size);

  initWith(2, arr, N);
  initWith(0, res, N);

  int blocks = 1;
  int threadsPerBlock = 16;
  int totalThreads = blocks * threadsPerBlock;

  float *ptemp;
  float *ttemp;
  hipMallocManaged(&ptemp, totalThreads * sizeof(float));
  hipMallocManaged(&ttemp, totalThreads * sizeof(float));

  prefixSum<<<blocks, threadsPerBlock>>>(arr, res, ptemp, ttemp, N);
  hipDeviceSynchronize();

  checkRes(arr, res, N);

  hipFree(arr);
  hipFree(res);
  hipFree(ttemp);
  hipFree(ptemp);
}