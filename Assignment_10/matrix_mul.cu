#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

__global__ void initMatrix(int *a, int val, int N){

    int col = blockIdx.x * blockDim.x + threadIdx.x;  //col ind
    int row = blockIdx.y * blockDim.y + threadIdx.y; //row ind

    if(row < N and col < N){

        c[row*N + col] = val;
    }
}

void check(int *a, int *b, int *c, int N){
    int sum;

    for(int i=0; i<N; i++){
        for(int j = 0; j<N; j++){
            sum = 0;
            for(int k = 0; k<N; k++){
                sum += a[i* N + k] * b[k*N + i];
            }

            if(sum != c[i*N + j])
                printf("wrong answer\n");
        }
    }

    printf("check successful\n");
}

__global__ void matrixMul(int *a, int *b, int *c, int N){

    int col = blockIdx.x * blockDim.x + threadIdx.x;  //col ind
    int row = blockIdx.y * blockDim.y + threadIdx.y; //row ind

    if(row < N and col < N){

        int sum = 0;
        for(int i=0; i<N; i++){
            sum += a[row * N + i] * b[i*N + col];
        }

        c[row*N + col] = sum;
    }

}

int main(){
    // Set our square matrix dimension (2%10 x 2*10 default)
    
    int N=1 << 10;
    size_t bytes = N * N * sizeof(int);

    // Allocate memory for our matrices
    int *a, *b, *c;
    
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    initMatrix<<<BLOCKS, THREADS>>>(a, 1, N);
    initMatrix<<<BLOCKS, THREADS>>>(b, 1, N);

       
    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    // Setup our kernel launch parameters
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    // Launch our kernel
    matrixMul<<<BLOCKS, THREADS>>>(a, b, c, N);
    hipDeviceSynchronize();

    check(a, b, c, N);

}
