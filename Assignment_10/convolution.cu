
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MASK_DIM 7

#define MASK_OFFSET (MASK_DIM / 2)

__constant__ int mask[7 * 7];

__global__ void convolution_2d(int *matrix, int *result, int N)
{
    // Calculate the global thread positions
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Starting index for calculation
    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;

    // Temp value for accumulating the result
    int temp = 0;

    // Iterate over all the rows
    for (int i = 0; i < MASK_DIM; i++)
    {
        // Go over each column
        for (int j = 0; j < MASK_DIM; j++)
        {
            // Range check for rows
            if ((start_r + i) >= 0 && (start_r + i) < N)
            {
                // Range check for columns
                if ((start_c + j) >= 0 && (start_c + j) < N)
                {
                    // Accumulate result
                    temp += matrix[(start_r + i) * N + (start_c + j)] * mask[i * MASK_DIM + j];
                }
            }
        }
    }

    // Write back the result
    result[row * N + col] = temp;
}

void init_matrix(int *m, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            m[n * i + j] = rand() % 100;
        }
    }
}

void verify_result(int *m, int *mask, int *result, int N)
{

    int temp;

    int offset_r;
    int offset_c;

    // Go over each row
    for (int i = 0; i < N; i++)
    {
        // Go over each column
        for (int j = 0; j < N; j++)
        {
            // Reset the temp variable
            temp = 0;

            // Go over each mask row
            for (int k = 0; k < MASK_DIM; k++)
            {
                // Update offset value for row
                offset_r = i - MASK_OFFSET + k;

                // Go over each mask column
                for (int l = 0; l < MASK_DIM; l++)
                {
                    // Update offset value for column
                    offset_c = j - MASK_OFFSET + l;

                    // Range checks if we are hanging off the matrix
                    if (offset_r >= 0 && offset_r < N)
                    {
                        if (offset_c >= 0 && offset_c < N)
                        {
                            // Accumulate partial results
                            temp += m[offset_r * N + offset_c] * mask[k * MASK_DIM + l];
                        }
                    }
                }
            }
            // Fail if the results don't match
            if (result[i * N + j] != temp)
            {
                printf("Check failed");
                return;
            }
        }
    }
}

int main()
{

    int N = 1 << 10; // 2^10

    size_t bytes_n = N * N * sizeof(int);
    size_t bytes_m = MASK_DIM * MASK_DIM * sizeof(int);

    int *matrix;
    int *result;
    int *h_mask;

    hipMallocManaged(&matrix, bytes_n);
    hipMallocManaged(&result, bytes_n);
    hipMallocManaged(&h_mask, bytes_m);

    init_matrix(matrix, N);
    init_matrix(mask, MASK_DIM);

    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    // Calculate grid dimensions
    int THREADS = 32;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    // Dimension launch arguments
    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(BLOCKS, BLOCKS);

    convolution_2d<<<grid_dim, block_dim>>>(matrix, result, N);

    verify_result(matrix, h_mask, result, N);

    printf("COMPLETED SUCCESSFULLY!");

    hipFree(matrix);
    hipFree(result);
    hipFree(h_mask);

    return 0;
}
